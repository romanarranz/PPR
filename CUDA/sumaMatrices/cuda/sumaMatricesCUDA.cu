#include "hip/hip_runtime.h"
#include <iostream>
using std::cout;
using std::cerr;
using std::endl;

#include "sumaMatrices.h"

// Error handling macro
#define CUDA_CHECK(call) \
    if((call) != hipSuccess) { \
        hipError_t err = hipGetLastError(); \
        cerr << "CUDA error calling \""#call"\", code is " << err << endl; }

// Device Code: Kernel 2D
__global__ void matAdd(float *A, float *B, float *C, int N){
    // Las matrices se recorren con la ordenacion de Fortran
    int j = blockIdx.x * blockDim.x + threadIdx.x; // indice filas
    int i = blockIdx.y * blockDim.y + threadIdx.y; // indice columnas
    int tid = (i * N) + j;

    if(i < N && j < N)
        C[tid] = A[tid] + B[tid];
}

void initMatrixes(float *h_A, float *h_B, float *h_C, int N){
    cout << "CPU: Inicializando los vectores..." << endl;
    for(int i = 0; i<N; i++){
        int row = N*i;
        for(int j = 0; j<N; j++){
            h_A[row+j] = row+j+1;
            h_B[row+j] =  row+j+2;
            h_C[row+j] = 0;
        }
    }
}

void matAddGPU(float *h_A, float *h_B, float *h_C, int N, int numBloques, int numThreadsBloque){
    unsigned int sizeMatrix = N * N;
    unsigned int memSize = sizeMatrix * sizeof(float);

    // GPU variables
    float * d_A = NULL;
    float * d_B = NULL;
    float * d_C = NULL;

    CUDA_CHECK(hipMalloc((void **)&d_A, memSize));
    CUDA_CHECK(hipMalloc((void **)&d_B, memSize));
    CUDA_CHECK(hipMalloc((void **)&d_C, memSize));

    cout << "CPU: Copiando las matrices de la CPU RAM a la GPU DRAM..." << endl;
    CUDA_CHECK(hipMemcpy(d_A, h_A, memSize, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, memSize, hipMemcpyHostToDevice));

    // Llamada a CUDA Kernel con N datos, usando 256 threads y bloque 1D, recordamos que los wraps de hebras se cogen de 32 hebras en 32
    // callback <<blocks_per_grid, thread_per_block>> (params);
    cout << "GPU: Calculando..." << endl;
    dim3 threadsPerBlock(numThreadsBloque,numThreadsBloque);
    dim3 numBlocks (numBloques, numThreadsBloque);
    matAdd<<<numBlocks, threadsPerBlock>>> (d_A, d_B, d_C, N);

    cout << "CPU: Copiando los resultados de la GPU DRAM a la CPU RAM..." << endl;
    CUDA_CHECK(hipMemcpy(h_C, d_C, memSize, hipMemcpyDeviceToHost));

    cout << "CPU: Liberando los datos de la GPU DRAM" << endl;
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));

    CUDA_CHECK(hipDeviceReset());
}
