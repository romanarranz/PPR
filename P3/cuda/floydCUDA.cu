#include "hip/hip_runtime.h"
#include <iostream>
using std::cout;
using std::cerr;
using std::endl;

#include <fstream>
#include <string.h>
#include <time.h>
#include "Graph.h"
#include "floyd.h"

#define CUDA_CHECK(call) \
    if((call) != hipSuccess) { \
        hipError_t err = hipGetLastError(); \
        cerr << "CUDA error calling \""#call"\", code is " << err << endl; }

__global__ void floyd1DKernel(int * M, const int nverts, const int k){
    short j = blockIdx.x * blockDim.x + threadIdx.x;    // indice filas
    short i = blockIdx.y;                               // indice columnas
    short tid = (i * nverts) + j;

    if(i!=j && i!=k && j!=k){
        if (i!=j && i!=k && j!=k) {
            short jk = (j*nverts) + k;
            short ki = (k*nverts) + i;
            short ij = (j*nverts) + i;
            int aux = M[jk]+M[ki];

            int vikj = min(aux, M[ij]);
            M[tid] = vikj;
        }
    }
}

// Kernel to update the Matrix at k-th iteration
__global__ void floyd2DKernel(int * M, const int nverts, const int k){
    short j = blockIdx.x * blockDim.x + threadIdx.x; // indice filas
    short i = blockIdx.y * blockDim.y + threadIdx.y; // indice columnas
    short tid = (i * nverts) + j;

    if(i < nverts && j < nverts){
        if (i!=j && i!=k && j!=k) {
            short jk = (j*nverts) + k;
            short ki = (k*nverts) + i;
            short ij = (j*nverts) + i;
            int aux = M[jk]+M[ki];

            int vikj = min(aux, M[ij]);
            M[tid] = vikj;
        }
    }
}

void floyd1DGPU(int *h_M, Graph g, int N, int numBloques, int numThreadsBloque){
    unsigned int sizeMatrix = N * N;
    unsigned int memSize = sizeMatrix * sizeof(int);

    // GPU variables
    int * d_M = NULL;

    CUDA_CHECK(hipMalloc((void **)&d_M, memSize));

    cout << "CPU: Copiando las matrices de la CPU RAM a la GPU DRAM..." << endl;
    CUDA_CHECK(hipMemcpy(d_M, h_M, memSize, hipMemcpyHostToDevice));

    cout << "GPU: Calculando..." << endl;
    for(int k = 0; k < N; k++){
        floyd1DKernel<<< numBloques, numThreadsBloque >>> (d_M, N, k);
    }

    cout << "CPU: Copiando los resultados de la GPU DRAM a la CPU RAM..." << endl;
    CUDA_CHECK(hipMemcpy(h_M, d_M, memSize, hipMemcpyDeviceToHost));
}

void floyd2DGPU(int *h_M, Graph g, int N, int numBloques, int numThreadsBloque){
    unsigned int sizeMatrix = N * N;
    unsigned int memSize = sizeMatrix * sizeof(int);

    // GPU variables
    int * d_M = NULL;

    CUDA_CHECK(hipMalloc((void **)&d_M, memSize));

    cout << "CPU: Copiando las matrices de la CPU RAM a la GPU DRAM..." << endl;
    CUDA_CHECK(hipMemcpy(d_M, h_M, memSize, hipMemcpyHostToDevice));

    cout << "GPU: Calculando..." << endl;
    for(int k = 0; k < N; k++){
        dim3 threadsPerBlock(numThreadsBloque,numThreadsBloque);
        dim3 numBlocks (numBloques, numThreadsBloque);
        floyd2DKernel<<< numBlocks, threadsPerBlock >>> (d_M, N, k);
    }

    cout << "CPU: Copiando los resultados de la GPU DRAM a la CPU RAM..." << endl;
    CUDA_CHECK(hipMemcpy(h_M, d_M, memSize, hipMemcpyDeviceToHost));
}
